#include "hip/hip_runtime.h"
%%writefile vecAdd_2.cu
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define DataType double
#define TPB 32
#define nstreams 4

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int t_n = gridDim.x * blockDim.x;
  while (tid < len)
  {
    out[tid] = in1[tid] + in2[tid];
    tid += t_n;
  }
}

//@@ Insert code to implement timer start
DataType cpuSecond(){
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

//@@ Insert code to implement timer stop

int main(int argc, char **argv) {

  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]);
  printf("The input length is %d\n", inputLength);

  //@@ Insert code below to allocate Host memory for input and output
  hipHostAlloc((void **)&hostInput1, inputLength*sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc((void **)&hostInput2, inputLength*sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc((void **)&hostOutput, inputLength*sizeof(DataType), hipHostMallocDefault);
  resultRef = (DataType *)malloc(sizeof(DataType)*inputLength);

  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  int i;
  for(i=0; i<inputLength; i++){
    hostInput1[i] = rand()%100;
    hostInput2[i] = rand()%100;
    resultRef[i] = hostInput1[i] + hostInput2[i];
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, sizeof(DataType)*inputLength);
  hipMalloc(&deviceInput2, sizeof(DataType)*inputLength);
  hipMalloc(&deviceOutput, sizeof(DataType)*inputLength);

  //@@ Insert code to below to Copy memory to the GPU here
  int S_seg = atoi(argv[2]);
  printf("the segment length is %d\n", S_seg);
  int circle1 = inputLength / (S_seg * nstreams);
  int circle2 = (inputLength - circle1 * S_seg * nstreams) / S_seg;
  int circle3 = inputLength - circle1 * S_seg * nstreams - circle2 * S_seg;
  int j;
  int c;
  const int streamSize = S_seg;
  const int streamBytes = streamSize * sizeof(double);
  hipStream_t stream[nstreams];
  for (i=0; i<nstreams; i++){
    hipStreamCreate(&stream[i]);
  }

  //@@ Initialize the 1D grid and block dimensions here
  //@@ Launch the GPU Kernel here
  DataType iStart = cpuSecond();
  for(i=0; i<circle1; i++){
    c = i * S_seg * nstreams;
    for(j=0; j<nstreams; j++){
      int offset = j * streamSize + c;
      hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], streamBytes, hipMemcpyHostToDevice, stream[j]);
      hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], streamBytes, hipMemcpyHostToDevice, stream[j]);
      vecAdd<<<((streamSize+TPB-1)/TPB), TPB, 0, stream[j]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], streamSize);
      hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset], streamBytes, hipMemcpyDeviceToHost, stream[j]);
      }
    hipDeviceSynchronize();
  }
  for(i=0; i<circle2; i++){
      int offset = i * streamSize + circle1 * S_seg * nstreams;
      hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);
      hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);
      vecAdd<<<((streamSize+TPB-1)/TPB), TPB, 0, stream[i]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], streamSize);
      hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);
      }
  int remainSize = circle3;
  int remainBytes = remainSize * sizeof(DataType);
  hipMemcpy(&deviceInput1[inputLength-remainSize], &hostInput1[inputLength-remainSize], remainBytes, hipMemcpyHostToDevice);
  hipMemcpy(&deviceInput2[inputLength-remainSize], &hostInput2[inputLength-remainSize], remainBytes, hipMemcpyHostToDevice);
  vecAdd<<<(remainSize+TPB-1)/TPB, TPB>>>(&deviceInput1[inputLength-remainSize], &deviceInput2[inputLength-remainSize], &deviceOutput[inputLength-remainSize], remainSize);
  hipMemcpy(&hostOutput[inputLength-remainSize], &deviceOutput[inputLength-remainSize], remainBytes, hipMemcpyDeviceToHost);

  DataType iElaps = cpuSecond() - iStart;

  for (i=0; i<nstreams; i++){
    hipStreamDestroy(stream[i]);
  }

  //@@ Copy the GPU memory back to the CPU here

  //@@ Insert code below to compare the output with the reference
  bool equal = false;
  int err;
  for (i=0; i<inputLength; i++){
    if(hostOutput[i] == resultRef[i]){
    equal = true;
  }
  else{
    equal = false;
    err = i;
    break;
  }
  }

  if(equal == true){
    printf("check passed! runtime: %f", iElaps);
  }
  else{
    printf("check error! error: %d", err);
  }


  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  hipHostFree(hostInput1);
  hipHostFree(hostInput2);
  hipHostFree(hostOutput);

  return 0;
}
